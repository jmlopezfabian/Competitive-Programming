//Programa que transfiere datos desde la memoria del host a la  memoria del device usando las funciones de manejo de memoria de CDUA
//Compilar: nvcc 03_mem.cu -o 01_mem.x                                                                                                                                    */                                                                                                                                                                      
#include <stdio.h>                                                                                                                                                      
#include <stdlib.h>                                                                                                                                                     
#include <hip/hip_runtime.h>                                                                                                                                               
#define N 10                                                                                                                                                            
#define forn(i,n) for(int i=0; i<n; i++)                                                                                                                                                                                                                                                                                                        
__global__ void imprime(float *arreglo, int a){                                                                                                                             
    int i=threadIdx.x + blockIdx.x*blockDim.x;                                                                                                                              
    if(i>=N*N) return ;                                                                                                                                                     
    arreglo[i]+=a;                                                                                                                                                          
    //printf("Arreglo %d = %f\n", i,arreglo[i]);                                                                                                                        
    }                                                                                                                                                                                                                                                                                                                                               
    // Función principal ejecutada en el host                                                                                                                               
    int main(int argc, char** argv){                                                                                                                                                                           
        // Declaración de variables                                                                                                                                             
        float *m_host;                                                                                                                                                          
        float *m_device;                                                                                                                                                                                                                                                                                                                                
        // Reservar memoria en el host                                                                                                                                          
        // m_host = (float *) malloc ( N * N * sizeof(float) );                                                                                                                 
        m_host = new float(N*N);                                                                                                                                                                                                                                                                                                                        
        // Reservar memoria en el device                                                                                                                                        
        hipMalloc( (void**) &m_device, N * N * sizeof(float) );                                                                                                                                                                                                                                                                                        
        // Inicializar la matriz                                                                                                                                                
        for (int i = 0; i < N * N; i++)                                                                                                                                             
        m_host[i] = (float) ( rand() % 10 );                                                                                                                                                                                                                                                                                                        
        //printf("Hola %f\n", m_host[100*100]);                                                                                                                                 
        // Copiar información al device                                                                                                                                         
        hipMemcpy(m_device, m_host, N * N * sizeof(float), hipMemcpyHostToDevice);                                                                                            
        int b=(N*N/1024) + 1;                                                                                                                                                   
        imprime<<<b,1024>>>(m_device,8);                                                                                                                                        
        hipMemcpy(m_host, m_device, N * N * sizeof(float), hipMemcpyDeviceToHost);                                                                                            
        forn(i,N*N)                                                                                                                                                                 
        printf("Arreglo %d = %f\n", i, m_host[i]);                                                                                                                                                                                                                                                                                                  
        // Liberar memoria                                                                                                                                                      
        hipFree( m_device );                                                                                                                                                   
        printf("\nPulsa INTRO para finalizar...");                                                                                                                              
        fflush(stdin);                                                                                                                                                          
        char tecla = getchar();                                                                                                                                                 
        return 0;                                                                                                                                                           } 


